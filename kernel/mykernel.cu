#include "hip/hip_runtime.h"
// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "particle.h"

//Adds two arrays
extern "C"
void runCudaPart(Particle* pArray);

__global__ void addAry( int * ary1, int * ary2 )
{
    int indx = threadIdx.x;
    ary1[ indx ] += ary2[ indx ];
}

// Main cuda function
void runCudaPart(Particle* pArray) {

    int ary1[32];
    int ary2[32];
    int res[32];

    for( int i=0 ; i<32 ; i++ )
    {
        ary1[i] = i;
        ary2[i] = 2*i;
        res[i]=0;
    }

    int* d_ary1;
    int* d_ary2;

    hipMalloc((void**)&d_ary1, 32*sizeof(int));
    hipMalloc((void**)&d_ary2, 32*sizeof(int));

    hipMemcpy((void*)d_ary1, (void*)ary1, 32*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_ary2, (void*)ary2, 32*sizeof(int), hipMemcpyHostToDevice);

    addAry<<<1,32>>>(d_ary1,d_ary2);

    hipMemcpy((void*)res, (void*)d_ary1, 32*sizeof(int), hipMemcpyDeviceToHost);
    for( int i=0 ; i<32 ; i++ )
        pArray[i].m_value = res[i];

    hipFree(d_ary1);
    hipFree(d_ary2);
}
