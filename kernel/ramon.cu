#include "hip/hip_runtime.h"
// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

#include "particlesystem.h"
#include "particle.h"

//Adds two arrays
extern "C"
void ramon(ParticleSystem& _ps, float * _vertices);

__global__ void blanquer(Particle* _particles)
{
    int index = threadIdx.x;
    _particles[index].m_pos_x = 0.9f;
    _particles[index].m_pos_y = 0.5f;
    _particles[index].m_pos_z = 0.0f;
}

// Main cuda function
void ramon(ParticleSystem& _ps, float * _vertices) {
    int size_of_ps = _ps.m_particles.size();

    Particle res[size_of_ps];
    Particle* d_particles;

    hipMalloc((void**)&d_particles, size_of_ps * sizeof(Particle));

    hipMemcpy((void**)d_particles, (void**)&_ps.m_particles[0], size_of_ps*sizeof(Particle), hipMemcpyHostToDevice);

    blanquer<<<1,size_of_ps>>>(d_particles);

    hipMemcpy((void*)res, (void*)d_particles, size_of_ps*sizeof(Particle), hipMemcpyDeviceToHost);

    for( int i = 0 ; i < size_of_ps ; i++)
    {
//        _vertices[i] = res[i].m_pos_x;
//        _vertices[i+1] = res[i].m_pos_x;
//        _vertices[i+2] = res[i].m_pos_x;
        //_ps.m_particles[i] = res[i];
        _vertices[i] = res[i].m_pos_x;
        _vertices[i+1] = res[i].m_pos_y;
        _vertices[i+2] = res[i].m_pos_z;
    }

    hipFree(d_particles);
}
