#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

__global__ void modifying_vbo_kernel(float *pos)
{
    unsigned int index = threadIdx.x;
    pos[index] = 0.9f;
}


// Main cuda function
void modifying_vbo(unsigned int VBO_id) {
    struct hipGraphicsResource *cuda_vbo_resource;
    void *d_vbo_buffer = NULL;
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, VBO_id, cudaGraphicsMapFlagsWriteDiscard);
    float* d_ptr;
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void**)&d_ptr, &num_bytes, cuda_vbo_resource);
    modifying_vbo_kernel<<<1, 9>>>(d_ptr);
    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}


