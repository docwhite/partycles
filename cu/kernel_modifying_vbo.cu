#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

__global__ void modifying_vbo_kernel(float *pos)
{
    unsigned int index = threadIdx.x;
    pos[index] = -0.9f;
}

void modifying_vbo(unsigned int VBO_id) {
    // Register the VBO so that we have a resource CUDA can understand and use
    struct hipGraphicsResource *cuda_vbo_resource;
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, VBO_id, cudaGraphicsMapFlagsWriteDiscard);

    // Initialize the device pointer we will use to access all the attribute array elements
    float* d_ptr;

    // Map the buffer to make it active for the kernel
    size_t num_bytes;
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);

    // Tells the right way to bind the device pointer to the correct resource
    hipGraphicsResourceGetMappedPointer((void**)&d_ptr, &num_bytes, cuda_vbo_resource);

    // Launch the kernel
    modifying_vbo_kernel<<<1, 9>>>(d_ptr);

    // Unmap the resources so OpenGL can now use the data to render
    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}


