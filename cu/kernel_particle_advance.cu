#include "hip/hip_runtime.h"
// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>

#include "particlesystem.h"
#include "particle.h"

__global__ void advance_particles_kernel(float* d_verts)
{
    int index = threadIdx.x;

    d_verts[index] = 0.8;
    d_verts[index+1] = 0.8;
    d_verts[index+2] = 0.8;
}

// Main cuda function
void advance_particles(ParticleSystem& _ps, float *_vertices) {
    int size_of_ps = _ps.m_particles.size();

    float  h_verts[3 * size_of_ps];
    float* d_verts = NULL;

    hipMalloc(&d_verts, 3 * size_of_ps * sizeof(float));

    advance_particles_kernel<<<1,size_of_ps>>>(d_verts);

    hipMemcpy(h_verts, d_verts, 3 * size_of_ps * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 3 * size_of_ps; i++) {
        _vertices[i] = h_verts[i];

    }

    hipFree(d_verts);
}
