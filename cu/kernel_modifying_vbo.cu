#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <iostream>

#include "particlesystem.h"

__global__ void modifying_vbo_kernel(float *d_vbo_positions, float *d_ps_positions, float *d_ps_velocities, int num_of_elements_per_array)
{
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < num_of_elements_per_array)
    {
    d_ps_positions[index] += d_ps_velocities[index];

    if (d_ps_positions[index] > 1.0f || d_ps_positions[index] < -1.0f)
        d_ps_velocities[index] = -d_ps_velocities[index];

    d_vbo_positions[index] = d_ps_positions[index];
    }
}

void modifying_vbo(unsigned int VBO_id, ParticleSystem* ps) {
    // Register the VBO so that we have a resource CUDA can understand and use
    struct hipGraphicsResource *cuda_vbo_resource;
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, VBO_id, cudaGraphicsMapFlagsWriteDiscard);

    // Initialize the device pointer we will use to access all the attribute array elements
    float* d_vbo_positions;
    float* d_ps_positions = thrust::raw_pointer_cast(&ps->positions[0]);
    float* d_ps_velocities = thrust::raw_pointer_cast(&ps->velocities[0]);

    // Map the buffer to make it active for the kernel
    size_t num_bytes;
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);

    // Tells the right way to bind the device pointer to the correct resource
    hipGraphicsResourceGetMappedPointer((void**)&d_vbo_positions, &num_bytes, cuda_vbo_resource);

    // std::cout << num_bytes / sizeof(float) << std::endl;

    unsigned int num_of_elements_per_array = 3 * ps->numParts;
    unsigned int block_size = 1024;
    unsigned int grid_size = num_of_elements_per_array / block_size + 1;

//    std::cout << "Number of elements in array: " << num_of_elements_per_array << std::endl;
//    std::cout << "Grid size: " << grid_size << std::endl;
//    std::cout << "Block size: " << block_size << std::endl;

    // Launch the kernel
    modifying_vbo_kernel<<<grid_size, block_size>>>(d_vbo_positions, d_ps_positions, d_ps_velocities, num_of_elements_per_array);


    // Unmap the resources so OpenGL can now use the data to render
    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}


